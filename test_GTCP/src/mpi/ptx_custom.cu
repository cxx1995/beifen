#include "hip/hip_runtime.h"
 __device__ __forceinline__ int custom_popc(unsigned int x)
{
	int ret;
    asm volatile ("{\n\t"
			".reg .u32 t1,t2;\n\t"
			"mov.u32 t1,%1;\n\t"
			"popc.b32 t2,t1;\n\t"
			"mov.u32 %0,t2;\n\t" 
			"}"
			: "=r"(ret) : "r"(x));
    return ret;
}

 __device__ __forceinline__ unsigned int select_gt_u32(unsigned int left, unsigned right)
{
	/*d = (sel == 1) ? a : b;*/
	unsigned int ret;
    asm volatile ("{\n\t"
			".reg .u32 a,b,c,d;\n\t"
			".reg .u32 l,r;\n\t"
			".reg .pred p;\n\t"
			"mov.u32 l,%1;\n\t"
			"mov.u32 r,%2;\n\t"
			"setp.u32.gt p,l,r;\n\t"
			"selp.u32 d,1,0,p;\n\t"
			"mov.u32 %0,d;\n\t"
			"}"
			: "=r"(ret) : "r"(left), "r"(right));
    return ret;
}

 __device__ __forceinline__ unsigned int select_eq_u32(unsigned int left, unsigned int right)
{
	/*d = (sel == 1) ? a : b;*/
	unsigned int ret;
    asm ("{\n\t"
			".reg .u32 d;\n\t"
			".reg .u32 l,r;\n\t"
			".reg .pred p;\n\t"
			"mov.u32 l,%1;\n\t"
			"mov.u32 r,%2;\n\t"
			"setp.u32.eq p,l,r;\n\t"
			"selp.u32 d,1,0,p;\n\t"
			"mov.u32 %0,d;\n\t" 
			"}"
			: "=r"(ret) : "r"(left), "r"(right));
    return ret;
}


 __device__ __forceinline__ unsigned int select_gt_f64(real left, real right)
{
	/*d = (sel == 1) ? a : b;*/
	unsigned int ret;
    asm ("{\n\t"
			".reg .u32 d;\n\t"
			".reg .f64 l,r;\n\t"
			".reg .pred p;\n\t"
			"mov.f64 l,%1;\n\t"
			"mov.f64 r,%2;\n\t"
			"setp.f64.gt p,l,r;\n\t"
			"selp.u32 d,1,0,p;\n\t"
			"mov.u32 %0,d;\n\t" 
			"}"
			: "=r"(ret) : "d"(left), "d"(right));
    return ret;
}

 __device__ __forceinline__ unsigned int select_eq_f64(real left, real right)
{
	/*d = (sel == 1) ? a : b;*/
	unsigned int ret;
    asm volatile ("{\n\t"
			".reg .u32 a,b,c,d;\n\t"
			".reg .f64 l,r;\n\t"
			".reg .pred p;\n\t"
			"mov.f64 l,%1;\n\t"
			"mov.f64 r,%2;\n\t"
			"setp.f64.eq p,l,r;\n\t"
			"selp.u32 d,1,0,p;\n\t"
			"mov.u32 %0,d;\n\t" 
			"}"
			: "=r"(ret) : "d"(left), "d"(right));
    return ret;
}


__device__ __forceinline__ int sorting_ballot(int sel, unsigned int count[3], int tidx, int tidy,unsigned int mask)
{
	int keep = (sel==0);
	int left = (sel==1);
	int right = (sel==2);
/*
	mask can be computed with but the compiler may not generate efficient code.
	asm ("{\n\t"
			".reg .u32 a;\n\t"
			"mov.u32 a, %lanemask_le;\n\t"
			"mov.u32 %0,a;\n\t" 
			"}"
			: "=r"(mask) : );

*/

	__shared__ unsigned int count_right_low, count_left_low,count_keep_low;
	unsigned int my_left_ballot = ballot(left);
	unsigned int my_right_ballot = ballot(right);
	unsigned int my_keep_ballot = ballot(keep);

	unsigned int my_count_right = custom_popc(my_right_ballot);
	unsigned int my_count_left = custom_popc(my_left_ballot);
	unsigned int my_count_keep = custom_popc(my_keep_ballot);
	int my_before_ballot = mask&(left*my_left_ballot+right*my_right_ballot+keep*my_keep_ballot);

	int order = custom_popc(my_before_ballot);
	if((tidx==0)&&(tidy==0)) {
		count_right_low = my_count_right;
		count_left_low = my_count_left;
		count_keep_low = my_count_keep;
	}
 	__syncthreads();
 	order += tidy*(count_right_low*right+count_left_low*left + count_keep_low*keep);

	if((tidx==31)&&(tidy==1)) {
		count[2] =  my_count_right + count_right_low;
		count[1] =  my_count_left + count_left_low;
		count[0] =  my_count_keep + count_keep_low;
	}
	return order;
}




/**
 * Enumeration of data movement cache modifiers.
 * adopted from back40 http://code.google.com/p/back40computing/
 */
enum CacheModifier {
	CG,
	CS, 
	CA,
	LU,
	CV,
	WB,
	WT

};

#if defined(__LP64__)
	#define _LP64_ true			
	// 64-bit register modifier for inlined asm
	#define _ASM_PTR_ "l"
#else
	#define _LP64_ false
	// 32-bit register modifier for inlined asm
	#define _ASM_PTR_ "r"
#endif


template <typename T, CacheModifier CACHE_MODIFIER> struct TunedTexLoad;

#define DEFINE_TEX_GLOBAL_LOAD(base_type, ptx_type, reg_mod)													\
		template <> struct TunedTexLoad<base_type, CG> {														\
			__device__ __forceinline__ static base_type Ld(const base_type* d_ptr) { 				\
				base_type val;																					\
				asm("ld.global.cg.nc."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));	\
				return val;																						\
			}																									\
		};																										\
		template <> struct TunedTexLoad<base_type, CS> {														\
			__device__ __forceinline__ static base_type Ld(const base_type* d_ptr) {\
				base_type val;																					\
				asm("ld.global.cs.nc."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));	\
				return val;																						\
			}																									\
		};																										\
		template <> struct TunedTexLoad<base_type, CA> {														\
			__device__ __forceinline__ static base_type Ld(const base_type* d_ptr) {\
				base_type val;																					\
				asm("ld.global.ca.nc."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));	\
				return val;																						\
			}																									\
		};					




DEFINE_TEX_GLOBAL_LOAD(double, f64, d)
DEFINE_TEX_GLOBAL_LOAD(float, f32, f)
DEFINE_TEX_GLOBAL_LOAD(short, s16, h)
DEFINE_TEX_GLOBAL_LOAD(int, s32, r)
DEFINE_TEX_GLOBAL_LOAD(long, s64, l)
DEFINE_TEX_GLOBAL_LOAD(unsigned short, u16, h)
DEFINE_TEX_GLOBAL_LOAD(unsigned int, u32, r)
DEFINE_TEX_GLOBAL_LOAD(unsigned long, u64, l)










template <typename T, CacheModifier CACHE_MODIFIER> struct TunedLoad;

#define DEFINE_BASE_GLOBAL_LOAD(base_type, ptx_type, reg_mod)													\
		template <> struct TunedLoad<base_type, CG> {															\
			__device__ __forceinline__ static void Ld(base_type &val, const base_type* d_ptr) {		\
				asm("ld.global.cg."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, CS> {															\
			__device__ __forceinline__ static void Ld(base_type &val, const base_type* d_ptr) {		\
				asm("ld.global.cs."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, LU> {															\
			__device__ __forceinline__ static void Ld(base_type &val, const base_type* d_ptr) {		\
				asm("ld.global.lu."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, CV> {															\
			__device__ __forceinline__ static void Ld(base_type &val, const base_type* d_ptr) {		\
				asm("ld.global.cv."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, CA> {															\
			__device__ __forceinline__ static void Ld(base_type &val, const base_type* d_ptr) {		\
				asm("ld.global.ca."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};					


#define DEFINE_GLOBAL_LOAD(base_type, dest_type, short_type, ptx_type, reg_mod)									\
		template <> struct TunedLoad<base_type, CG> {															\
			__device__ __forceinline__ static void Ld(dest_type &val, const base_type* d_ptr) {		\
				asm("ld.global.cg."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, CS> {															\
			__device__ __forceinline__ static void Ld(dest_type &val, const base_type* d_ptr) {		\
				asm("ld.global.cs."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, LU> {															\
			__device__ __forceinline__ static void Ld(dest_type &val, const base_type* d_ptr) {		\
				asm("ld.global.lu."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, CV> {															\
			__device__ __forceinline__ static void Ld(dest_type &val, const base_type* d_ptr) {		\
				asm("ld.global.cv."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\
		template <> struct TunedLoad<base_type, CA> {															\
			__device__ __forceinline__ static void Ld(dest_type &val, const base_type* d_ptr) {		\
				asm("ld.global.ca."#ptx_type" %0, [%1];" : "="#reg_mod(val) : _ASM_PTR_(d_ptr));		\
			}																									\
		};																										\

	// Cache-modified loads for built-in structures
//	DEFINE_GLOBAL_LOAD(char, signed char, char, s8, r)
//	DEFINE_BASE_GLOBAL_LOAD(signed char, s8, r)		
	DEFINE_GLOBAL_LOAD(short, short, short, s16, h)
	DEFINE_GLOBAL_LOAD(int, int, int, s32, r)
	DEFINE_GLOBAL_LOAD(long, long, long, s64, l)
	DEFINE_GLOBAL_LOAD(long long, long long, longlong, s64, l)
//	DEFINE_GLOBAL_LOAD(unsigned char, unsigned char, uchar, u8, r)
	DEFINE_GLOBAL_LOAD(unsigned short, unsigned short, ushort, u16, h)
	DEFINE_GLOBAL_LOAD(unsigned int, unsigned int, uint, u32, r)
	DEFINE_GLOBAL_LOAD(unsigned long, unsigned long, ulong, u64, l)
	DEFINE_GLOBAL_LOAD(unsigned long long, unsigned long long, ulonglong, u64, l)
	DEFINE_GLOBAL_LOAD(float, float, float, f32, f)
	DEFINE_BASE_GLOBAL_LOAD(double, f64, d)


template <typename T, CacheModifier CACHE_MODIFIER> struct TunedStore;

#define DEFINE_BASE_GLOBAL_STORE(base_type, ptx_type, reg_mod)													\
		template <> struct TunedStore<base_type, CG> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.cg."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																										\
		template <> struct TunedStore<base_type, CS> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.cs."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																										\
		template <> struct TunedStore<base_type, WT> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.wt."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																										\
		template <> struct TunedStore<base_type, WB> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.wb."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																																								
#define DEFINE_GLOBAL_STORE(base_type, dest_type, short_type, ptx_type, reg_mod)								\
		template <> struct TunedStore<base_type, CG> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.cg."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																										\
		template <> struct TunedStore<base_type, CS> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.cs."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																										\
		template <> struct TunedStore<base_type, WT> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.wt."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																										\
		template <> struct TunedStore<base_type, WB> {															\
			__device__ __forceinline__ static void St(const base_type &val, base_type* d_ptr) {		\
				asm("st.global.wb."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));			\
			}																									\
		};																										\

	// Cache-modified stores for built-in structures
//	DEFINE_GLOBAL_STORE(char, signed char, char, s8, r)
//	DEFINE_BASE_GLOBAL_STORE(signed char, s8, r)		
	DEFINE_GLOBAL_STORE(short, short, short, s16, h)
	DEFINE_GLOBAL_STORE(int, int, int, s32, r)
	DEFINE_GLOBAL_STORE(long, long, long, s64, l)
	DEFINE_GLOBAL_STORE(long long, long long, longlong, s64, l)
//	DEFINE_GLOBAL_STORE(unsigned char, unsigned char, uchar, u8, r)
	DEFINE_GLOBAL_STORE(unsigned short, unsigned short, ushort, u16, h)
	DEFINE_GLOBAL_STORE(unsigned int, unsigned int, uint, u32, r)
	DEFINE_GLOBAL_STORE(unsigned long, unsigned long, ulong, u64, l)
	DEFINE_GLOBAL_STORE(unsigned long long, unsigned long long, ulonglong, u64, l)
	DEFINE_GLOBAL_STORE(float, float, float, f32, f)
	DEFINE_BASE_GLOBAL_STORE(double, f64, d)	

#define DEFINE_POPC(base_type, ptx_type, reg_mod)			\
	 __device__ __forceinline__ int popc_instr_##ptx_type(base_type x) {	\
		int ret;				\
	    asm ("popc."#ptx_type" %0,%1;\n\t": "=r"(ret) : #reg_mod(x)); \
    	return ret; \
	} 

DEFINE_POPC(unsigned int,b32,r)
DEFINE_POPC(unsigned long long,b64,l)




#define DEFINE_EXTRACT_BITS(base_type, ptx_type,reg_mod)					\
__device__ __forceinline__ static void ExtractKeyBits(int bit_start, int num_bits, base_type &bits, const base_type &source) { 	\
	asm("bfe."#ptx_type"  %0, %1, %2, %3;" : "="#reg_mod(bits) : #reg_mod(source), "r"(bit_start), "r"(num_bits)); 	\
}

DEFINE_EXTRACT_BITS(unsigned int, u32, r);
DEFINE_EXTRACT_BITS(unsigned long long, u64, l);


#define DEFINE_REVERSE_BITS(base_type, ptx_type,reg_mod)			\
__device__ __forceinline__ void mirror_bitmap(base_type &reversed, base_type input) \
{ 																			\
	asm("brev."#ptx_type" %0, %1;" : "="#reg_mod(reversed): #reg_mod(input));	\
} 

DEFINE_REVERSE_BITS(unsigned int, b32, r);
DEFINE_REVERSE_BITS(unsigned long long, b64, l);

/*
	We define the bits involved in sorting as the bits that differs across keys
	To find that we reduce the key using and operation. Ones are considered same so no need to sort for these bits.
	Then We do reduction over the negation of the variable. Ones are also not involved.
	Finally we or values in the first reduction with values in the second and then we negate that to define bits for sorting.


	If we have 16 bits for comparison, we hope by paying these two extra reductions we will save in the number of ballot voting later on.
 

*/
enum reductionOp {
	AND, OR, XOR, /*.b32 only*/
	add, /* .u32, .s32, .f32, .u64 */
	INC, DEC, /* .u32 only */
	MIN, MAX /* .u32, .s32 */
};

template <typename T, reductionOp REDUCTION_OP> struct OptimizedReduction;



#define DEFINE_REDUCE_BITMAP(base_type, ptx_type,reg_mod, op, ptx_op)								\
	template <> struct OptimizedReduction<base_type, op> { 											\
		__device__ __forceinline__ static void Barrier(const base_type &val, base_type* d_ptr) {		\
			asm("red."#ptx_op"."#ptx_type" [%0], %1;" : : _ASM_PTR_(d_ptr), #reg_mod(val));	\
		}																							\
	};																								\

DEFINE_REDUCE_BITMAP(unsigned int, b32, r, AND, and);
DEFINE_REDUCE_BITMAP(unsigned int, b32, r, OR, or);
DEFINE_REDUCE_BITMAP(unsigned int, b32, r, XOR, xor);


